#include "hip/hip_runtime.h"
#include <_Time.h>
#include <_File.h>
#include <_String.h>
#include <random>
#include <hip/hip_runtime.h>
#include <>
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>
#include <hiprand/hiprand_kernel.h>


constexpr unsigned int BlockHeights[4]
{
	4,
	4,
	8,
	16
};

//sizes: 128, 256, 512, 1024
const dim3 Ising3DBlockSize[4]
{
	{1, 32, 1},//blockHeight == 4
	{1, 64, 1},//blockHeight == 4
	{4, 64, 1},//blockHeight == 8
	{16, 64, 1},//blockHeight == 16
};

const dim3 Ising3DThreadSize[4]
{
	{2, 128, 1},
	{4, 256, 1},
	{8, 128, 1},
	{16, 64, 1},
};

const dim3 InitGridBlockSize[4]
{
	{2, 1, 1},
	{64, 1, 1},//blockHeight == 4
	{256, 1, 1},//blockHeight == 8
	{1024, 1, 1},//blockHeight == 16
};

constexpr unsigned long long ReductionMBlockSize[4]{ 32, 64, 256, 1024 };

const unsigned int RandomSize[4]
{
	2048,
	65536,//blockHeight == 4
	262144,//blockHeight == 8
	1048576//blockHeight == 16
};

const dim3 RandomBlockSize[4]
{
	{2, 1, 1},
	{64, 1, 1},//blockHeight == 4
	{256, 1, 1},//blockHeight == 8
	{256, 1, 1},//blockHeight == 16
};

constexpr unsigned long long ReductionBlockSize[4]{ 32, 256, 1024, 1024 };

constexpr __device__ unsigned int getSubWidth(unsigned int dim)
{
	if (dim > 256)return 256 / (dim / 256) + 2;
	else return dim;
}
constexpr __device__ unsigned int getBlockHeight(unsigned int dim)
{
	if (dim > 512)return 16;
	else if (dim > 256)return 8;
	else return 4;
}
constexpr __device__ unsigned int getBlockSize(unsigned int dim)
{
	if (dim > 128)return 1024;
	return 256;
}

//test for size of 1024
__global__ void initRandom(hiprandState* states, unsigned int seed, unsigned long long nStates)
{
	int id(blockIdx.x * blockDim.x + threadIdx.x);
	unsigned int gridSize(blockDim.x * gridDim.x);
	while (id < nStates)
	{
		hiprand_init(seed, id, 0, states + id);
		id += gridSize;
	}
}
__global__ void initGrid(unsigned long long* grid, hiprandState* states, unsigned long long N)
{
	unsigned long long stateIdx(threadIdx.x + 1024 * blockIdx.x);
	unsigned long long idx(stateIdx);
	unsigned long long gridSize(gridDim.x * 1024);
	hiprandState state(states[stateIdx]);
	while (idx < N)
	{
		unsigned long long a(hiprand(&state));
		a <<= 32;
		a |= hiprand(&state);
		grid[idx] = a;
		idx += gridSize;
	}
	states[stateIdx] = state;
}
template<unsigned long long dim>__global__ void ising3d(unsigned long long* grid, float H, float T, int step, hiprandState* states)
{
	constexpr unsigned int dimM1(dim - 1);
	constexpr unsigned int bandWidth(256 / (dim / 256));
	constexpr unsigned int subGridBandWidth(getSubWidth(dim));
	constexpr unsigned int rowLength(dim / 64);
	constexpr unsigned long long layerSize(dim * dim / 64);
	__shared__ unsigned long long subGrid[3][subGridBandWidth][rowLength];
	__shared__ float expDeltaE[2][7];
	constexpr unsigned int blockHeight(getBlockHeight(dim));
	unsigned int c0(blockIdx.y * blockHeight);
	unsigned int c0e(c0 + blockHeight);
	unsigned int idx(0);
	int randIdx(blockIdx.y * layerSize + (threadIdx.y + blockIdx.x * bandWidth) * rowLength + threadIdx.x);
	hiprandState state(states[randIdx]);
	//build table
	if (threadIdx.x < 2 && threadIdx.y < 7)
		expDeltaE[threadIdx.x][threadIdx.y] = expf((2 - 4 * int(threadIdx.x)) * (H + float(2 * threadIdx.y) - 6) / T);
	//gather the extra rows, for dim == 1024, 512
	if constexpr (dim > 256)
	{
		if (threadIdx.y == 0)
			subGrid[1][0][threadIdx.x] = grid[c0 * layerSize + ((blockIdx.x * bandWidth + dimM1) & dimM1) * rowLength + threadIdx.x];
		else if (threadIdx.y == 1)
			subGrid[1][bandWidth + 1][threadIdx.x] = grid[c0 * layerSize + (((blockIdx.x + 1) * bandWidth) & dimM1) * rowLength + threadIdx.x];
		//gather the main part
		subGrid[0][threadIdx.y + 1][threadIdx.x] = grid[((c0 + dimM1) & dimM1) * layerSize + (blockIdx.x * bandWidth + threadIdx.y) * rowLength + threadIdx.x];
		subGrid[1][threadIdx.y + 1][threadIdx.x] = grid[c0 * layerSize + (blockIdx.x * bandWidth + threadIdx.y) * rowLength + threadIdx.x];
	}
	else
	{
		//gather the main part
		subGrid[0][threadIdx.y][threadIdx.x] = grid[((c0 + dimM1) & dimM1) * layerSize + threadIdx.y * rowLength + threadIdx.x];
		subGrid[1][threadIdx.y][threadIdx.x] = grid[c0 * layerSize + threadIdx.y * rowLength + threadIdx.x];
	}
	for (; c0 < c0e; ++c0)
	{
		idx = (idx + 1) % 3;
		if constexpr (dim > 256)
		{
			if (c0 < c0e - 1)
			{
				if (threadIdx.y == 0)
					subGrid[(idx + 1) % 3][0][threadIdx.x] = grid[((c0 + 1) & dimM1) * layerSize + ((blockIdx.x * bandWidth + dimM1) & dimM1) * rowLength + threadIdx.x];
				else if (threadIdx.y == 1)
					subGrid[(idx + 1) % 3][bandWidth + 1][threadIdx.x] = grid[((c0 + 1) & dimM1) * layerSize + (((blockIdx.x + 1) * bandWidth) & dimM1) * rowLength + threadIdx.x];
			}
		}
		if constexpr (dim > 256)
			subGrid[(idx + 1) % 3][threadIdx.y + 1][threadIdx.x] = grid[((c0 + 1) & dimM1) * layerSize + (blockIdx.x * bandWidth + threadIdx.y) * rowLength + threadIdx.x];
		else
			subGrid[(idx + 1) % 3][threadIdx.y][threadIdx.x] = grid[((c0 + 1) & dimM1) * layerSize + threadIdx.y * rowLength + threadIdx.x];
		__syncthreads();
		unsigned long long center;
		unsigned long long nearby[4];
		int s0, s1;
		if constexpr (dim > 256)
		{
			nearby[0] = subGrid[(idx + 2) % 3][threadIdx.y + 1][threadIdx.x];
			nearby[1] = subGrid[idx][threadIdx.y][threadIdx.x];
			center = subGrid[idx][threadIdx.y + 1][threadIdx.x];
			nearby[2] = subGrid[idx][threadIdx.y + 2][threadIdx.x];
			nearby[3] = subGrid[(idx + 1) % 3][threadIdx.y + 1][threadIdx.x];
		}
		else
		{
			constexpr unsigned int bdwM1(bandWidth - 1);
			nearby[0] = subGrid[(idx + 2) % 3][threadIdx.y][threadIdx.x];
			nearby[1] = subGrid[idx][(threadIdx.y + bdwM1) & bdwM1][threadIdx.x];
			center = subGrid[idx][threadIdx.y][threadIdx.x];
			nearby[2] = subGrid[idx][(threadIdx.y + 1) & bdwM1][threadIdx.x];
			nearby[3] = subGrid[(idx + 1) % 3][threadIdx.y][threadIdx.x];
		}
#define get(x, y) (((x) >> (y)) & 1)
#define set(x) (center ^= (1llu << (x)))
		int stepNow((step + c0 + threadIdx.y) & 1);
		constexpr unsigned int rlM1(rowLength - 1);
		if (stepNow == 0)
		{
			if constexpr (dim > 256)s0 = get(subGrid[idx][threadIdx.y + 1][(threadIdx.x + rlM1) & rlM1], 63);
			else s0 = get(subGrid[idx][threadIdx.y][(threadIdx.x + rlM1) & rlM1], 63);
		}
		else s0 = center & 1;
		for (int c1(stepNow); c1 < 64; c1 += 2)
		{
			if (c1 < 63)s1 = get(center, c1 + 1);
			else
			{
				if constexpr (dim > 256)s1 = get(subGrid[idx][threadIdx.y + 1][(threadIdx.x + 1) & rlM1], 0);
				else s1 = get(subGrid[idx][threadIdx.y][(threadIdx.x + 1) & rlM1], 0);
			}
			s0 += s1;
			for (int c2(0); c2 < 4; ++c2)s0 += get(nearby[c2], c1);
			int ss(get(center, c1));
			if (hiprand_uniform(&state) < expDeltaE[ss][s0])set(c1);
			s0 = s1;
		}
		if constexpr (dim > 256)grid[c0 * layerSize + (blockIdx.x * bandWidth + threadIdx.y) * rowLength + threadIdx.x] = center;
		else grid[c0 * layerSize + threadIdx.y * rowLength + threadIdx.x] = center;
	}
	states[randIdx] = state;
#undef get
#undef set
}
template<unsigned long long dim>__global__ void ising3d(unsigned long long* grid, float H, float T, long long* MList, long long* EList, int step, hiprandState* states)
{
	constexpr unsigned int dimM1(dim - 1);
	constexpr unsigned int bandWidth(256 / (dim / 256));
	constexpr unsigned int subGridBandWidth(getSubWidth(dim));
	constexpr unsigned int rowLength(dim / 64);
	constexpr unsigned long long layerSize(dim * dim / 64);
	__shared__ unsigned long long subGrid[3][subGridBandWidth][rowLength];
	__shared__ float expDeltaE[2][7];
	constexpr unsigned int blockHeight(getBlockHeight(dim));
	unsigned int c0(blockIdx.y * blockHeight);
	unsigned int c0e(c0 + blockHeight);
	unsigned int idx(0);
	int randIdx(blockIdx.y * layerSize + (threadIdx.y + blockIdx.x * bandWidth) * rowLength + threadIdx.x);
	int sumM(0), sumE(0);
	hiprandState state(states[randIdx]);
	//build table
	if (threadIdx.x < 2 && threadIdx.y < 7)
		expDeltaE[threadIdx.x][threadIdx.y] = expf((2 - 4 * int(threadIdx.x)) * (H + float(2 * threadIdx.y) - 6) / T);
	//gather the extra rows, for dim == 1024, 512
	if constexpr (dim > 256)
	{
		if (threadIdx.y == 0)
			subGrid[1][0][threadIdx.x] = grid[c0 * layerSize + ((blockIdx.x * bandWidth + dimM1) & dimM1) * rowLength + threadIdx.x];
		else if (threadIdx.y == 1)
			subGrid[1][bandWidth + 1][threadIdx.x] = grid[c0 * layerSize + (((blockIdx.x + 1) * bandWidth) & dimM1) * rowLength + threadIdx.x];
		//gather the main part
		subGrid[0][threadIdx.y + 1][threadIdx.x] = grid[((c0 + dimM1) & dimM1) * layerSize + (blockIdx.x * bandWidth + threadIdx.y) * rowLength + threadIdx.x];
		subGrid[1][threadIdx.y + 1][threadIdx.x] = grid[c0 * layerSize + (blockIdx.x * bandWidth + threadIdx.y) * rowLength + threadIdx.x];
	}
	else
	{
		//gather the main part
		subGrid[0][threadIdx.y][threadIdx.x] = grid[((c0 + dimM1) & dimM1) * layerSize + threadIdx.y * rowLength + threadIdx.x];
		subGrid[1][threadIdx.y][threadIdx.x] = grid[c0 * layerSize + threadIdx.y * rowLength + threadIdx.x];
	}
	for (; c0 < c0e; ++c0)
	{
		idx = (idx + 1) % 3;
		if constexpr (dim > 256)
		{
			if (c0 < c0e - 1)
			{
				if (threadIdx.y == 0)
					subGrid[(idx + 1) % 3][0][threadIdx.x] = grid[((c0 + 1) & dimM1) * layerSize + ((blockIdx.x * bandWidth + dimM1) & dimM1) * rowLength + threadIdx.x];
				else if (threadIdx.y == 1)
					subGrid[(idx + 1) % 3][bandWidth + 1][threadIdx.x] = grid[((c0 + 1) & dimM1) * layerSize + (((blockIdx.x + 1) * bandWidth) & dimM1) * rowLength + threadIdx.x];
			}
		}
		if constexpr (dim > 256)
			subGrid[(idx + 1) % 3][threadIdx.y + 1][threadIdx.x] = grid[((c0 + 1) & dimM1) * layerSize + (blockIdx.x * bandWidth + threadIdx.y) * rowLength + threadIdx.x];
		else
			subGrid[(idx + 1) % 3][threadIdx.y][threadIdx.x] = grid[((c0 + 1) & dimM1) * layerSize + threadIdx.y * rowLength + threadIdx.x];
		__syncthreads();
		unsigned long long center;
		unsigned long long nearby[4];
		int s0, s1;
		if constexpr (dim > 256)
		{
			nearby[0] = subGrid[(idx + 2) % 3][threadIdx.y + 1][threadIdx.x];
			nearby[1] = subGrid[idx][threadIdx.y][threadIdx.x];
			center = subGrid[idx][threadIdx.y + 1][threadIdx.x];
			nearby[2] = subGrid[idx][threadIdx.y + 2][threadIdx.x];
			nearby[3] = subGrid[(idx + 1) % 3][threadIdx.y + 1][threadIdx.x];
		}
		else
		{
			constexpr unsigned int bdwM1(bandWidth - 1);
			nearby[0] = subGrid[(idx + 2) % 3][threadIdx.y][threadIdx.x];
			nearby[1] = subGrid[idx][(threadIdx.y + bdwM1) & bdwM1][threadIdx.x];
			center = subGrid[idx][threadIdx.y][threadIdx.x];
			nearby[2] = subGrid[idx][(threadIdx.y + 1) & bdwM1][threadIdx.x];
			nearby[3] = subGrid[(idx + 1) % 3][threadIdx.y][threadIdx.x];
		}
#define get(x, y) (((x) >> (y)) & 1)
#define set(x) (center ^= (1llu << (x)))
		int stepNow((step + c0 + threadIdx.y) & 1);
		constexpr unsigned int rlM1(rowLength - 1);
		if (stepNow == 0)
		{
			if constexpr (dim > 256)s0 = get(subGrid[idx][threadIdx.y + 1][(threadIdx.x + rlM1) & rlM1], 63);
			else s0 = get(subGrid[idx][threadIdx.y][(threadIdx.x + rlM1) & rlM1], 63);
		}
		else s0 = center & 1;
		for (int c1(stepNow); c1 < 64; c1 += 2)
		{
			if (c1 < 63)s1 = get(center, c1 + 1);
			else
			{
				if constexpr (dim > 256)s1 = get(subGrid[idx][threadIdx.y + 1][(threadIdx.x + 1) & rlM1], 0);
				else s1 = get(subGrid[idx][threadIdx.y][(threadIdx.x + 1) & rlM1], 0);
			}
			s0 += s1;
			for (int c2(0); c2 < 4; ++c2)s0 += get(nearby[c2], c1);
			int ss(get(center, c1));
			if (hiprand_uniform(&state) < expDeltaE[ss][s0])set(c1);
			sumM += ss + s1;
			sumE -= (2 * ss - 1) * (2 * s0 - 6);
			s0 = s1;
		}
		if constexpr (dim > 256)grid[c0 * layerSize + (blockIdx.x * bandWidth + threadIdx.y) * rowLength + threadIdx.x] = center;
		else grid[c0 * layerSize + threadIdx.y * rowLength + threadIdx.x] = center;
	}
	states[randIdx] = state;
#undef get
#undef set
	constexpr unsigned int blockSize(getBlockSize(dim));
	__shared__ int gatherM[blockSize], gatherE[blockSize];
	unsigned int id(threadIdx.x + threadIdx.y * rowLength);
	gatherM[id] = sumM;
	gatherE[id] = sumE;
	__syncthreads();
	if constexpr (blockSize >= 1024)if (id < 512) { gatherM[id] += gatherM[id + 512]; gatherE[id] += gatherE[id + 512]; __syncthreads(); }
	if constexpr (blockSize >= 512)if (id < 256) { gatherM[id] += gatherM[id + 256]; gatherE[id] += gatherE[id + 256]; __syncthreads(); }
	if constexpr (blockSize >= 256)if (id < 128) { gatherM[id] += gatherM[id + 128]; gatherE[id] += gatherE[id + 128]; __syncthreads(); }
	if constexpr (blockSize >= 128)if (id < 64) { gatherM[id] += gatherM[id + 64]; gatherE[id] += gatherE[id + 64]; __syncthreads(); }
	if (id < 32)//blockSize must be greater than 64...
	{
		gatherM[id] += gatherM[id + 32];
		gatherE[id] += gatherE[id + 32]; __syncthreads();
		gatherM[id] += gatherM[id + 16];
		gatherE[id] += gatherE[id + 16]; __syncthreads();
		gatherM[id] += gatherM[id + 8];
		gatherE[id] += gatherE[id + 8]; __syncthreads();
		gatherM[id] += gatherM[id + 4];
		gatherE[id] += gatherE[id + 4]; __syncthreads();
		gatherM[id] += gatherM[id + 2];
		gatherE[id] += gatherE[id + 2]; __syncthreads();
		gatherM[id] += gatherM[id + 1];
		gatherE[id] += gatherE[id + 1]; __syncthreads();
	}
	if (id == 0)
	{
		unsigned int blockId(blockIdx.x + blockIdx.y * gridDim.x);
		MList[blockId] = gatherM[0];
		EList[blockId] = gatherE[0];
	}
}
template<class T, unsigned long long blockSize>__device__ void warpReduce(volatile T* sdata, unsigned int tid)
{
	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
	if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
	if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
	if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
	if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}
template<unsigned long long blockSize>__global__ void isingReduction(unsigned long long* a, unsigned long long* b, unsigned long long N)
{
	unsigned int tid(threadIdx.x);
	unsigned long long i(blockIdx.x * blockSize + tid);
	unsigned long long gridSize(blockSize * gridDim.x);
	unsigned long long ans(0);
	while (i < N)
	{
		unsigned long long gg = a[i];
		unsigned int n;
		asm("popc.b64 %0, %1;": "=r"(n) : "l"(gg));
		ans += n;
		i += gridSize;
	}
	__shared__ unsigned long long sdata[blockSize];//must fix it size and don't use extern!!!
	sdata[tid] = ans;
	__syncthreads();
	if constexpr (blockSize == 1024)
	{
		if (tid < 512)
			sdata[tid] += sdata[tid + 512];
		__syncthreads();
	}
	if constexpr (blockSize >= 512)
	{
		if (tid < 256)
			sdata[tid] += sdata[tid + 256];
		__syncthreads();
	}
	if constexpr (blockSize >= 256)
	{
		if (tid < 128)
			sdata[tid] += sdata[tid + 128];
		__syncthreads();
	}
	if constexpr (blockSize >= 128)
	{
		if (tid < 64)
			sdata[tid] += sdata[tid + 64];
		__syncthreads();
	}
	if (tid < 32)warpReduce<unsigned long long, blockSize>(sdata, tid);
	if (tid == 0)b[blockIdx.x] = sdata[0];
}
template<class T, unsigned long long blockSize>__global__ void reduction(T* a, T* b, T N)
{
	unsigned int tid(threadIdx.x);
	unsigned long long i(blockIdx.x * blockSize + tid);
	unsigned long long gridSize(blockSize * gridDim.x);
	T ans(0);
	while (i < N)
	{
		ans += a[i];
		i += gridSize;
	}
	__shared__ T sdata[blockSize];//must fix it size and don't use extern!!!
	sdata[tid] = ans;
	__syncthreads();
	if constexpr (blockSize == 1024)
	{
		if (tid < 512)
			sdata[tid] += sdata[tid + 512];
		__syncthreads();
	}
	if constexpr (blockSize >= 512)
	{
		if (tid < 256)
			sdata[tid] += sdata[tid + 256];
		__syncthreads();
	}
	if constexpr (blockSize >= 256)
	{
		if (tid < 128)
			sdata[tid] += sdata[tid + 128];
		__syncthreads();
	}
	if constexpr (blockSize >= 128)
	{
		if (tid < 64)
			sdata[tid] += sdata[tid + 64];
		__syncthreads();
	}
	if (tid < 32)warpReduce<T, blockSize>(sdata, tid);
	if (tid == 0)b[blockIdx.x] = sdata[0];
}

constexpr unsigned int chooseIdx(unsigned long long dim)
{
	if (dim == 1024)return 3;
	if (dim == 512)return 2;
	if (dim == 256)return 1;
	if (dim == 128)return 0;
}

int main()
{
	File file("./");
	float T(1.1);
	float H0(1e-1);
	float H1(0);
	::printf("T: %f\nH0: %f\nH1: %f\n", T, H0, H1);
	Timer timer;
	std::mt19937 mt(time(0));
	std::uniform_int_distribution<unsigned long long>rd;
	File& sf(file.findInThis("states.bin"));
	bool statesNeeded(false);
	if (&sf == nullptr)statesNeeded = true;
	constexpr size_t dim(256);
	constexpr size_t spinNum(dim * dim * dim);
	constexpr size_t gridSize(spinNum / 8);
	constexpr size_t gridNum(spinNum / 64);
	constexpr size_t layerSize(dim * dim / 8);
	constexpr size_t layerNum(dim * dim / 64);
	constexpr unsigned int idx(chooseIdx(dim));
	size_t statesSize(sizeof(hiprandState) * (layerNum * dim) / BlockHeights[idx]);
	::printf("stateSize: %llu\n", statesSize);
	size_t sumSize(ReductionBlockSize[idx] * sizeof(unsigned long long));
	void* states(::malloc(statesSize));
	unsigned long long* debugBuffer((unsigned long long*)::malloc(layerSize));
	hiprandState* statesDevice;
	//unsigned long long* grid((unsigned long long*)::malloc(gridSize));
	unsigned long long* gridDevice;
	unsigned long long* sumDevice;
	void* sumSumDevice;
	long long* sumMDevice, * sumEDevice;
	//for (unsigned long long c0(0); c0 < gridSize / sizeof(unsigned long long); ++c0)grid[c0] = rd(mt);
	hipMalloc(&gridDevice, gridSize);
	hipMalloc(&statesDevice, statesSize);
	hipMalloc(&sumDevice, sumSize);
	hipMalloc(&sumSumDevice, sizeof(unsigned long long));
	hipMalloc(&sumMDevice, ReductionMBlockSize[idx] * sizeof(long long));
	hipMalloc(&sumEDevice, ReductionMBlockSize[idx] * sizeof(long long));

	if (statesNeeded)
	{
		hipDeviceSynchronize();
		timer.begin();
		initRandom << <RandomBlockSize[idx], 1024, 0, 0 >> > (statesDevice, rd(mt), RandomSize[idx]);
		hipDeviceSynchronize();
		timer.end();
		timer.print("Generate states: ");
	}
	else
	{
		timer.begin();
		Vector<unsigned char>ss(sf.readBinary());
		hipMemcpy(statesDevice, ss.data, statesSize, hipMemcpyHostToDevice);
		timer.end();
		timer.print("Read states: ");
	}

	for (unsigned long long c0(0); c0 < 1; ++c0)
	{
		hipDeviceSynchronize();
		//timer.begin();
		initGrid << <InitGridBlockSize[idx], 1024 >> > (gridDevice, statesDevice, gridNum);
		hipDeviceSynchronize();
		//timer.end();
		//timer.print("Init Grid: ");
		timer.begin();
		float dH(H0 - H1);
		for (unsigned long long c1(0); c1 < 2000; ++c1)
		{
			float H;
			if (c1 < 100)H = H0;
			else H = H1;
			if (c1 % 50 == 0)
			{
				isingReduction <1024> << <ReductionBlockSize[idx], 1024 >> > (gridDevice, sumDevice, spinNum / 64);
				reduction<unsigned long long, ReductionBlockSize[idx]> << <1, ReductionBlockSize[idx] >> > (sumDevice, (unsigned long long*)sumSumDevice, ReductionBlockSize[idx]);
				unsigned long long sumAnswer;
				hipMemcpy(&sumAnswer, sumSumDevice, sizeof(unsigned long long), hipMemcpyDeviceToHost);
				::printf("%d: %f, %lf\n", c1, H, double(2 * sumAnswer) / spinNum - 1);
			}
			ising3d<dim> << <Ising3DBlockSize[idx], Ising3DThreadSize[idx] >> > (gridDevice, H, T, 0, statesDevice);
			ising3d<dim> << <Ising3DBlockSize[idx], Ising3DThreadSize[idx] >> > (gridDevice, H, T, 1, statesDevice);
		}
		hipDeviceSynchronize();
		timer.end();
		timer.print("Ising3D: ");

		//timer.begin();
		isingReduction <1024> << <ReductionBlockSize[idx], 1024 >> > (gridDevice, sumDevice, spinNum / 64);
		reduction<unsigned long long, ReductionBlockSize[idx]> << <1, ReductionBlockSize[idx] >> > (sumDevice, (unsigned long long*)sumSumDevice, ReductionBlockSize[idx]);
		unsigned long long sumAnswer;
		hipMemcpy(&sumAnswer, sumSumDevice, sizeof(unsigned long long), hipMemcpyDeviceToHost);
		//timer.end();
		//timer.print("Reduce: ");
		double M(double(2 * sumAnswer) / spinNum - 1);
		::printf("Average M:\t%lf\n", M);
		/*if (abs(M) < 0.99)
		{
			for (unsigned long long cc(128); cc < 129; ++cc)
			{
				hipMemcpy(debugBuffer, gridDevice + (cc & (dim - 1)) * layerNum, layerSize, hipMemcpyDeviceToHost);
				for (unsigned long long c1(0); c1 < dim; ++c1)
				{
					for (unsigned long long c2(0); c2 < dim / 64; ++c2)
						for (unsigned long long c3(0); c3 < 64; ++c3)
							::printf("%d", (debugBuffer[dim * c1 / 64 + c2] >> c3) & 1);
					::printf("\n");
				}
				::printf("\n");
				::printf("\n");
			}
		}*/

		long long MLL, ELL;
		double E;
		ising3d<dim> << <Ising3DBlockSize[idx], Ising3DThreadSize[idx] >> > (gridDevice, H1, T, 0, statesDevice);
		ising3d<dim> << <Ising3DBlockSize[idx], Ising3DThreadSize[idx] >> > (gridDevice, H1, T, sumMDevice, sumEDevice, 1, statesDevice);
		reduction<long long, ReductionMBlockSize[idx]> << <1, ReductionMBlockSize[idx] >> > (sumMDevice, (long long*)sumSumDevice, ReductionMBlockSize[idx]);
		hipMemcpy(&MLL, sumSumDevice, sizeof(long long), hipMemcpyDeviceToHost);
		reduction<long long, ReductionMBlockSize[idx]> << <1, ReductionMBlockSize[idx] >> > (sumEDevice, (long long*)sumSumDevice, ReductionMBlockSize[idx]);
		hipMemcpy(&ELL, sumSumDevice, sizeof(long long), hipMemcpyDeviceToHost);
		M = double(2 * MLL) / spinNum - 1;
		E = double(ELL) / spinNum - M * H1;
		::printf("M: %lf\tE: %lf\n", M, E);
	}

	timer.begin();
	hipMemcpy(states, statesDevice, statesSize, hipMemcpyDeviceToHost);
	file.createBinary("states.bin", states, statesSize);
	timer.end();
	timer.print("Store states: ");

	hipFree(gridDevice);
	hipFree(statesDevice);
	hipFree(sumDevice);
	hipFree(sumSumDevice);
	hipFree(sumMDevice);
	hipFree(sumEDevice);
	//free(grid);
	free(states);
	free(debugBuffer);
}