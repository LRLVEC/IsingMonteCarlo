#include "hip/hip_runtime.h"
#include <_Time.h>
#include <_File.h>
#include <_String.h>
#include <random>
#include <hip/hip_runtime.h>
#include <>
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>
#include <hiprand/hiprand_kernel.h>


constexpr unsigned int blockHeight = 4;

//sizes: 128, 256, 512, 1024
const dim3 Ising3DBlockSize[4]
{
	{1, 8,1},
	{1, 64,1},//blockHeight == 4
	{4, 32,1},
	{16, 64,1},
};

const dim3 Ising3DThreadSize[4]
{
	{8, 128, 1},
	{4, 256, 1},
	{8, 128, 1},
	{16, 64, 1},
};

const dim3 InitGridBlockSize[4]
{
	{2, 1, 1},
	{64, 1, 1},//blockHeight == 4
	{128, 1, 1},//blockHeight == 16
	{1024, 1, 1},//blockHeight == 16
};

const unsigned int RandomSize[4]{ 2048,16384,131072,1048576 };

const dim3 RandomBlockSize[4]
{
	{2, 1, 1},
	{64, 1, 1},//blockHeight == 4
	{128, 1, 1},//blockHeight == 16
	{256, 1, 1},//blockHeight == 16
};

constexpr unsigned int ReductionBlockSize[4]{ 32,256, 1024, 1024 };

constexpr __device__ unsigned int getSubWidth(unsigned int dim)
{
	if (dim > 256)return 256 / (dim / 256) + 2;
	else return dim;
}

//test for size of 1024
__global__ void initRandom(hiprandState* states, unsigned int seed, unsigned long long nStates)
{
	int id(blockIdx.x * blockDim.x + threadIdx.x);
	unsigned int gridSize(blockDim.x * gridDim.x);
	while (id < nStates)
	{
		hiprand_init(seed, id, 0, states + id);
		id += gridSize;
	}
}
__global__ void initGrid(unsigned long long* grid, hiprandState* states, unsigned long long N)
{
	unsigned long long stateIdx(threadIdx.x + 1024 * blockIdx.x);
	unsigned long long idx(stateIdx);
	unsigned long long gridSize(gridDim.x * 1024);
	hiprandState state(states[stateIdx]);
	while (idx < N)
	{
		unsigned long long a(hiprand(&state));
		a <<= 32;
		a |= hiprand(&state);
		grid[idx] = a;
		idx += gridSize;
	}
	states[stateIdx] = state;
}
template<unsigned long long dim>__global__ void ising3d(unsigned long long* grid, float H, float T, int step, hiprandState* states)
{
	constexpr unsigned int dimM1(dim - 1);
	constexpr unsigned int bandWidth(256 / (dim / 256));
	constexpr unsigned int subGridBandWidth(getSubWidth(dim));
	constexpr unsigned int rowLength(dim / 64);
	constexpr unsigned long long layerSize(dim * dim / 64);
	__shared__ unsigned long long subGrid[3][subGridBandWidth][rowLength];
	__shared__ float table[2][7];
	unsigned int c0(blockIdx.y * blockHeight);
	unsigned int c0e(c0 + blockHeight);
	unsigned int idx(0);
	int randIdx(blockIdx.y * layerSize + (threadIdx.y + blockIdx.x * bandWidth) * rowLength + threadIdx.x);
	hiprandState state(states[randIdx]);
	//build table
	if (threadIdx.x < 2 && threadIdx.y < 7)
		table[threadIdx.x][threadIdx.y] = expf((2 - 4 * int(threadIdx.x)) * (H + float(2 * threadIdx.y) - 6) / T);
	//gather the extra rows, for dim == 1024, 512
	if constexpr (dim > 256)
	{
		if (threadIdx.y == 0)
			subGrid[1][0][threadIdx.x] = grid[c0 * layerSize + ((blockIdx.x * bandWidth + dimM1) & dimM1) * rowLength + threadIdx.x];
		else if (threadIdx.y == 1)
			subGrid[1][bandWidth + 1][threadIdx.x] = grid[c0 * layerSize + (((blockIdx.x + 1) * bandWidth) & dimM1) * rowLength + threadIdx.x];
		//gather the main part
		subGrid[0][threadIdx.y + 1][threadIdx.x] = grid[((c0 + dimM1) & dimM1) * layerSize + (blockIdx.x * bandWidth + threadIdx.y) * rowLength + threadIdx.x];
		subGrid[1][threadIdx.y + 1][threadIdx.x] = grid[c0 * layerSize + (blockIdx.x * bandWidth + threadIdx.y) * rowLength + threadIdx.x];
	}
	else
	{
		//gather the main part
		subGrid[0][threadIdx.y][threadIdx.x] = grid[((c0 + dimM1) & dimM1) * layerSize + threadIdx.y * rowLength + threadIdx.x];
		subGrid[1][threadIdx.y][threadIdx.x] = grid[c0 * layerSize + threadIdx.y * rowLength + threadIdx.x];
	}
	for (; c0 < c0e; ++c0)
	{
		idx = (idx + 1) % 3;
		if constexpr (dim > 256)
		{
			if (c0 < c0e - 1)
			{
				if (threadIdx.y == 0)
					subGrid[(idx + 1) % 3][0][threadIdx.x] = grid[((c0 + 1) & dimM1) * layerSize + ((blockIdx.x * bandWidth + dimM1) & dimM1) * rowLength + threadIdx.x];
				else if (threadIdx.y == 1)
					subGrid[(idx + 1) % 3][bandWidth + 1][threadIdx.x] = grid[((c0 + 1) & dimM1) * layerSize + (((blockIdx.x + 1) * bandWidth) & dimM1) * rowLength + threadIdx.x];
			}
		}
		if constexpr (dim > 256)
			subGrid[(idx + 1) % 3][threadIdx.y + 1][threadIdx.x] = grid[((c0 + 1) & dimM1) * layerSize + (blockIdx.x * bandWidth + threadIdx.y) * rowLength + threadIdx.x];
		else
			subGrid[(idx + 1) % 3][threadIdx.y][threadIdx.x] = grid[((c0 + 1) & dimM1) * layerSize + threadIdx.y * rowLength + threadIdx.x];
		__syncthreads();
		unsigned long long center;
		unsigned long long nearby[4];
		int s0, s1;
		if constexpr (dim > 256)
		{
			nearby[0] = subGrid[(idx + 2) % 3][threadIdx.y + 1][threadIdx.x];
			nearby[1] = subGrid[idx][threadIdx.y][threadIdx.x];
			center = subGrid[idx][threadIdx.y + 1][threadIdx.x];
			nearby[2] = subGrid[idx][threadIdx.y + 2][threadIdx.x];
			nearby[3] = subGrid[(idx + 1) % 3][threadIdx.y + 1][threadIdx.x];
		}
		else
		{
			constexpr unsigned int bdwM1(bandWidth - 1);
			nearby[0] = subGrid[(idx + 2) % 3][threadIdx.y][threadIdx.x];
			nearby[1] = subGrid[idx][(threadIdx.y + bdwM1) & bdwM1][threadIdx.x];
			center = subGrid[idx][threadIdx.y][threadIdx.x];
			nearby[2] = subGrid[idx][(threadIdx.y + 1) & bdwM1][threadIdx.x];
			nearby[3] = subGrid[(idx + 1) % 3][threadIdx.y][threadIdx.x];
		}
#define get(x, y) (((x) >> (y)) & 1)
#define set(x) (center ^= (1llu << (x)))
		int stepNow((step + c0 + threadIdx.y) & 1);
		constexpr unsigned int rlM1(rowLength - 1);
		if (stepNow == 0)
			s0 = get(subGrid[idx][threadIdx.y + 1][(threadIdx.x + rlM1) & rlM1], 63);
		else
			s0 = center & 1;
		for (int c1(stepNow); c1 < 64; c1 += 2)
		{
			if (c1 < 63)
				s1 = get(center, c1 + 1);
			else
				s1 = get(subGrid[idx][threadIdx.y + 1][(threadIdx.x + 1) & rlM1], 0);
			s0 += s1;
			for (int c2(0); c2 < 4; ++c2)
				s0 += get(nearby[c2], c1);
			int ss(get(center, c1));
			if (hiprand_uniform(&state) < table[ss][s0])
				set(c1);
			s0 = s1;
		}
		if constexpr (dim > 256)
			grid[c0 * layerSize + (blockIdx.x * bandWidth + threadIdx.y) * rowLength + threadIdx.x] = center;
		else
			grid[c0 * layerSize + threadIdx.y * rowLength + threadIdx.x] = center;
	}
	states[randIdx] = state;
#undef get
#undef set
}
//template<unsigned long long dim>__global__ void ising3d(unsigned long long* grid, float H, float T, int step, hiprandState* states, float* MList, float* EList)
//{
//	constexpr unsigned int dimM1(dim - 1);
//	constexpr unsigned int bandWidth(256 / (dim / 256));
//	constexpr unsigned int subGridBandWidth(getSubWidth(dim));
//	constexpr unsigned int rowLength(dim / 64);
//	constexpr unsigned long long layerSize(dim * dim / 64);
//	__shared__ unsigned long long subGrid[3][subGridBandWidth][rowLength];
//	__shared__ float table[2][7];
//	unsigned int c0(blockIdx.y * blockHeight);
//	unsigned int c0e(c0 + blockHeight);
//	unsigned int idx(0);
//	int randIdx(blockIdx.y * layerSize + (threadIdx.y + blockIdx.x * bandWidth) * rowLength + threadIdx.x);
//	hiprandState state(states[randIdx]);
//	//build table
//	if (threadIdx.x < 2 && threadIdx.y < 7)
//		table[threadIdx.x][threadIdx.y] = expf((2 - 4 * int(threadIdx.x)) * (H + float(2 * threadIdx.y) - 6) / T);
//	//gather the extra rows, for dim == 1024, 512
//	if constexpr (dim > 256)
//	{
//		if (threadIdx.y == 0)
//			subGrid[1][0][threadIdx.x] = grid[c0 * layerSize + ((blockIdx.x * bandWidth + dimM1) & dimM1) * rowLength + threadIdx.x];
//		else if (threadIdx.y == 1)
//			subGrid[1][bandWidth + 1][threadIdx.x] = grid[c0 * layerSize + (((blockIdx.x + 1) * bandWidth) & dimM1) * rowLength + threadIdx.x];
//		//gather the main part
//		subGrid[0][threadIdx.y + 1][threadIdx.x] = grid[((c0 + dimM1) & dimM1) * layerSize + (blockIdx.x * bandWidth + threadIdx.y) * rowLength + threadIdx.x];
//		subGrid[1][threadIdx.y + 1][threadIdx.x] = grid[c0 * layerSize + (blockIdx.x * bandWidth + threadIdx.y) * rowLength + threadIdx.x];
//	}
//	else
//	{
//		//gather the main part
//		subGrid[0][threadIdx.y][threadIdx.x] = grid[((c0 + dimM1) & dimM1) * layerSize + threadIdx.y * rowLength + threadIdx.x];
//		subGrid[1][threadIdx.y][threadIdx.x] = grid[c0 * layerSize + threadIdx.y * rowLength + threadIdx.x];
//	}
//	for (; c0 < c0e; ++c0)
//	{
//		idx = (idx + 1) % 3;
//		if constexpr (dim > 256)
//		{
//			if (c0 < c0e - 1)
//			{
//				if (threadIdx.y == 0)
//					subGrid[(idx + 1) % 3][0][threadIdx.x] = grid[((c0 + 1) & dimM1) * layerSize + ((blockIdx.x * bandWidth + dimM1) & dimM1) * rowLength + threadIdx.x];
//				else if (threadIdx.y == 1)
//					subGrid[(idx + 1) % 3][bandWidth + 1][threadIdx.x] = grid[((c0 + 1) & dimM1) * layerSize + (((blockIdx.x + 1) * bandWidth) & dimM1) * rowLength + threadIdx.x];
//			}
//		}
//		if constexpr (dim > 256)
//			subGrid[(idx + 1) % 3][threadIdx.y + 1][threadIdx.x] = grid[((c0 + 1) & dimM1) * layerSize + (blockIdx.x * bandWidth + threadIdx.y) * rowLength + threadIdx.x];
//		else
//			subGrid[(idx + 1) % 3][threadIdx.y][threadIdx.x] = grid[((c0 + 1) & dimM1) * layerSize + threadIdx.y * rowLength + threadIdx.x];
//		__syncthreads();
//		unsigned long long center;
//		unsigned long long nearby[4];
//		int s0, s1;
//		if constexpr (dim > 256)
//		{
//			nearby[0] = subGrid[(idx + 2) % 3][threadIdx.y + 1][threadIdx.x];
//			nearby[1] = subGrid[idx][threadIdx.y][threadIdx.x];
//			center = subGrid[idx][threadIdx.y + 1][threadIdx.x];
//			nearby[2] = subGrid[idx][threadIdx.y + 2][threadIdx.x];
//			nearby[3] = subGrid[(idx + 1) % 3][threadIdx.y + 1][threadIdx.x];
//		}
//		else
//		{
//			constexpr unsigned int bdwM1(bandWidth - 1);
//			nearby[0] = subGrid[(idx + 2) % 3][threadIdx.y][threadIdx.x];
//			nearby[1] = subGrid[idx][(threadIdx.y + bdwM1) & bdwM1][threadIdx.x];
//			center = subGrid[idx][threadIdx.y][threadIdx.x];
//			nearby[2] = subGrid[idx][(threadIdx.y + 1) & bdwM1][threadIdx.x];
//			nearby[3] = subGrid[(idx + 1) % 3][threadIdx.y][threadIdx.x];
//		}
//#define get(x, y) (((x) >> (y)) & 1)
//#define set(x) (center ^= (1llu << (x)))
//		int stepNow((step + c0 + threadIdx.y) & 1);
//		constexpr unsigned int rlM1(rowLength - 1);
//		if (stepNow == 0)
//			s0 = get(subGrid[idx][threadIdx.y + 1][(threadIdx.x + rlM1) & rlM1], 63);
//		else
//			s0 = center & 1;
//		for (int c1(stepNow); c1 < 64; c1 += 2)
//		{
//			if (c1 < 63)
//				s1 = get(center, c1 + 1);
//			else
//				s1 = get(subGrid[idx][threadIdx.y + 1][(threadIdx.x + 1) & rlM1], 0);
//			s0 += s1;
//			for (int c2(0); c2 < 4; ++c2)
//				s0 += get(nearby[c2], c1);
//			int ss(get(center, c1));
//			if (hiprand_uniform(&state) < table[ss][s0])
//				set(c1);
//			s0 = s1;
//		}
//		if constexpr (dim > 256)
//			grid[c0 * layerSize + (blockIdx.x * bandWidth + threadIdx.y) * rowLength + threadIdx.x] = center;
//		else
//			grid[c0 * layerSize + threadIdx.y * rowLength + threadIdx.x] = center;
//	}
//	states[randIdx] = state;
//#undef get
//#undef set
//}
template<unsigned long long blockSize>__device__ void warpReduce(volatile unsigned long long* sdata, unsigned int tid)
{
	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
	if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
	if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
	if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
	if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}
template<unsigned long long blockSize>__global__ void isingReduction(unsigned long long* a, unsigned long long* b, unsigned long long N)
{
	unsigned int tid(threadIdx.x);
	unsigned long long i(blockIdx.x * blockSize + tid);
	unsigned long long gridSize(blockSize * gridDim.x);
	unsigned long long ans(0);
	while (i < N)
	{
		unsigned long long gg = a[i];
		unsigned int n;
		asm("popc.b64 %0, %1;": "=r"(n) : "l"(gg));
		ans += n;
		i += gridSize;
	}
	__shared__ unsigned long long sdata[blockSize];//must fix it size and don't use extern!!!
	sdata[tid] = ans;
	__syncthreads();
	if constexpr (blockSize == 1024)
	{
		if (tid < 512)
			sdata[tid] += sdata[tid + 512];
		__syncthreads();
	}
	if constexpr (blockSize >= 512)
	{
		if (tid < 256)
			sdata[tid] += sdata[tid + 256];
		__syncthreads();
	}
	if constexpr (blockSize >= 256)
	{
		if (tid < 128)
			sdata[tid] += sdata[tid + 128];
		__syncthreads();
	}
	if constexpr (blockSize >= 128)
	{
		if (tid < 64)
			sdata[tid] += sdata[tid + 64];
		__syncthreads();
	}
	if (tid < 32)warpReduce<blockSize>(sdata, tid);
	if (tid == 0)b[blockIdx.x] = sdata[0];
}
template<unsigned long long blockSize>__global__ void reduction(unsigned long long* a, unsigned long long* b, unsigned long long N)
{
	unsigned int tid(threadIdx.x);
	unsigned long long i(blockIdx.x * blockSize + tid);
	unsigned long long gridSize(blockSize * gridDim.x);
	unsigned long long ans(0);
	while (i < N)
	{
		ans += a[i];
		i += gridSize;
	}
	__shared__ unsigned long long sdata[blockSize];//must fix it size and don't use extern!!!
	sdata[tid] = ans;
	__syncthreads();
	if constexpr (blockSize == 1024)
	{
		if (tid < 512)
			sdata[tid] += sdata[tid + 512];
		__syncthreads();
	}
	if constexpr (blockSize >= 512)
	{
		if (tid < 256)
			sdata[tid] += sdata[tid + 256];
		__syncthreads();
	}
	if constexpr (blockSize >= 256)
	{
		if (tid < 128)
			sdata[tid] += sdata[tid + 128];
		__syncthreads();
	}
	if constexpr (blockSize >= 128)
	{
		if (tid < 64)
			sdata[tid] += sdata[tid + 64];
		__syncthreads();
	}
	if (tid < 32)warpReduce<blockSize>(sdata, tid);
	if (tid == 0)b[blockIdx.x] = sdata[0];
}

constexpr unsigned int chooseIdx(unsigned long long dim)
{
	if (dim == 1024)return 3;
	if (dim == 512)return 2;
	if (dim == 256)return 1;
	if (dim == 128)return 0;
}

int main()
{
	File file("./");
	float T(1);
	float H(0);
	Timer timer;
	std::mt19937 mt(time(0));
	std::uniform_int_distribution<unsigned long long>rd;
	File& sf(file.findInThis("states.bin"));
	bool statesNeeded(false);
	if (&sf == nullptr)statesNeeded = true;
	constexpr size_t dim(256);
	constexpr size_t spinNum(dim * dim * dim);
	constexpr size_t gridSize(spinNum / 8);
	constexpr size_t gridNum(spinNum / 64);
	constexpr size_t layerSize(dim * dim / 8);
	constexpr size_t layerNum(dim * dim / 64);
	constexpr unsigned int idx(chooseIdx(dim));
	size_t statesSize(sizeof(hiprandState) * (layerNum * dim) / blockHeight);
	::printf("stateSize: %llu\n", statesSize);
	size_t sumSize(ReductionBlockSize[idx] * sizeof(unsigned long long));
	void* states(::malloc(statesSize));
	unsigned long long* debugBuffer((unsigned long long*)::malloc(layerSize));
	hiprandState* statesDevice;
	//unsigned long long* grid((unsigned long long*)::malloc(gridSize));
	unsigned long long* gridDevice;
	unsigned long long* sumDevice;
	unsigned long long* sumSumDevice;
	//for (unsigned long long c0(0); c0 < gridSize / sizeof(unsigned long long); ++c0)grid[c0] = rd(mt);
	hipMalloc(&gridDevice, gridSize);
	hipMalloc(&statesDevice, statesSize);
	hipMalloc(&sumDevice, sumSize);
	hipMalloc(&sumSumDevice, sizeof(unsigned long long));

	if (statesNeeded)
	{
		hipDeviceSynchronize();
		timer.begin();
		initRandom << <RandomBlockSize[idx], 1024, 0, 0 >> > (statesDevice, rd(mt), RandomSize[idx]);
		hipDeviceSynchronize();
		timer.end();
		timer.print("Generate states: ");
	}
	else
	{
		timer.begin();
		Vector<unsigned char>ss(sf.readBinary());
		hipMemcpy(statesDevice, ss.data, statesSize, hipMemcpyHostToDevice);
		timer.end();
		timer.print("Read states: ");
	}

	for (unsigned long long c0(0); c0 < 1; ++c0)
	{
		//hipMemcpy(gridDevice, grid, gridSize, hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		timer.begin();
		initGrid << <InitGridBlockSize[idx], 1024 >> > (gridDevice, statesDevice, gridNum);
		hipDeviceSynchronize();
		timer.end();
		timer.print("Init Grid: ");

		timer.begin();
		for (unsigned long long c1(0); c1 < 100000; ++c1)
		{
			ising3d<dim> << <Ising3DBlockSize[idx], Ising3DThreadSize[idx] >> > (gridDevice, H, T, 0, statesDevice);
			ising3d<dim> << <Ising3DBlockSize[idx], Ising3DThreadSize[idx] >> > (gridDevice, H, T, 1, statesDevice);
		}
		hipDeviceSynchronize();
		timer.end();
		timer.print("Ising3D: ");

		timer.begin();
		isingReduction <1024> << <ReductionBlockSize[idx], 1024 >> > (gridDevice, sumDevice, spinNum / 64);
		reduction<ReductionBlockSize[idx]> << <1, ReductionBlockSize[idx] >> > (sumDevice, sumSumDevice, ReductionBlockSize[idx]);
		/*for (unsigned long long cc(0); cc < 1; ++cc)
		{
			hipMemcpy(debugBuffer, gridDevice + (cc & (dim - 1)) * layerNum, layerSize, hipMemcpyDeviceToHost);
			for (unsigned long long c1(0); c1 < dim; ++c1)
			{
				for (unsigned long long c2(0); c2 < dim / 64; ++c2)
					for (unsigned long long c3(0); c3 < 64; ++c3)
						::printf("%d", (debugBuffer[dim * c1 / 64 + c2] >> c3) & 1);
				::printf("\n");
			}
			::printf("\n");
			::printf("\n");
		}*/
		unsigned long long sumAnswer;
		hipMemcpy(&sumAnswer, sumSumDevice, sizeof(unsigned long long), hipMemcpyDeviceToHost);
		timer.end();
		timer.print("Reduce: ");
		::printf("Average M:\t%lf\n", double(2 * sumAnswer) / spinNum - 1);
	}

	timer.begin();
	hipMemcpy(states, statesDevice, statesSize, hipMemcpyDeviceToHost);
	file.createBinary("states.bin", states, statesSize);
	timer.end();
	timer.print("Store states: ");

	hipFree(gridDevice);
	hipFree(statesDevice);
	hipFree(sumDevice);
	hipFree(sumSumDevice);
	//free(grid);
	free(states);
	free(debugBuffer);
}