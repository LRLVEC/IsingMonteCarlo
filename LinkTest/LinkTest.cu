#include "hip/hip_runtime.h"
#include "LinkTest.cuh"
#include "hip/hip_runtime.h"
#include ""


__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

void addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;

	hipSetDevice(0);

	hipMalloc((void**)&dev_c, size * sizeof(int));
	hipMalloc((void**)&dev_a, size * sizeof(int));
	hipMalloc((void**)&dev_b, size * sizeof(int));
	hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	addKernel << <1, size >> > (dev_c, dev_a, dev_b);
	hipDeviceSynchronize();
	hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
}

template<unsigned int D>void AHH<D>::set()
{
	gg[0] = 1;
}